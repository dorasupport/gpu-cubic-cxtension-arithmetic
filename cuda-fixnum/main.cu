#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstring>
#include <cassert>

#include "fixnum/warp_fixnum.cu"
#include "array/fixnum_array.h"
#include "functions/modexp.cu"
#include "functions/multi_modexp.cu"
#include "modnum/modnum_monty_redc.cu"
#include "modnum/modnum_monty_cios.cu"
#include "mnt4_g1.cu"
#include "mnt4_g2.cu"
#include "mnt6_g1.cu"
#include "mnt6_g2.cu"

using namespace std;
using namespace cuFIXNUM;
using namespace MNT_G;
int BLOCK_NUM = 4096;
#define MNT_SIZE (96)
#define PARALLEL_SIGMA

// mnt4_q
const uint8_t mnt4_modulus[MNT_SIZE] = {1,128,94,36,222,99,144,94,159,17,221,44,82,84,157,227,240,37,196,154,113,16,136,99,164,84,114,118,233,204,90,104,56,126,83,203,165,13,15,184,157,5,24,242,118,231,23,177,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0};

// mnt6_q
uint8_t mnt6_modulus[MNT_SIZE] = {1,0,0,64,226,118,7,217,79,58,161,15,23,153,160,78,151,87,0,63,188,129,195,214,164,58,153,52,118,249,223,185,54,38,33,41,148,202,235,62,155,169,89,200,40,92,108,178,157,247,90,161,217,36,209,153,141,237,160,232,37,185,253,7,115,216,151,108,249,232,183,94,237,175,143,91,80,151,249,183,173,205,226,238,34,144,34,16,17,196,146,45,198,196,1,0};

// mnt4 a, for calc 2p, not use now
uint8_t mnt4_a[MNT_SIZE] = {0x84,0xde,0xb8,0xb3,0x57,0xd9,0x51,0x31,0xd,0x8d,0x6,0xb4,0x8c,0x63,0x9a,0x23,0x5d,0xae,0x28,0x9a,0x41,0xc9,0x87,0x2f,0x3,0x6c,0x11,0x8f,0x33,0x30,0xb1,0xf2,0xde,0x2e,0x11,0x42,0x28,0x39,0x4d,0xda,0xd1,0x3a,0x06,0x9e,0x15,0x9b,0x1e,0x3c,0xb2,0xa,0x67,0x26,0x6e,0x77,0x18,0x64,0xc4,0x14,0xe0,0xa5,0x05,0x86,0x16,0xb3,0x42,0x4c,0x19,0xfb,0x97,0x93,0xe9,0x80,0x18,0xd1,0xcb,0x70,0xb6,0xfd,0x48,0x1f,0x2a,0x43,0xf3,0x3f,0x66,0xbf,0x8a,0x2a,0x85,0xc4,0x91,0x3d,0x8f,0xf6};

template< typename fixnum >
__device__ void dump(fixnum n, int size) {
#if 0
	for (int i = 0; i < size; i++) {
		printf("DUMP [%d] %x\n", i, fixnum::get(n, i));
	}
#endif
	printf("dump [%d]=\%x\n", threadIdx.x, fixnum::get(n, threadIdx.x));
}

template< typename fixnum >
struct mnt4g1_pq_plus {
    __device__ void operator()(fixnum mod, fixnum x1, fixnum y1, fixnum z1, fixnum x2, fixnum y2, fixnum z2, fixnum &x3, fixnum &y3, fixnum &z3) {
        typedef mnt4_g1<fixnum> mnt4g1;
        mnt4g1::pq_plus(mod, x1, y1, z1, x2, y2, z2, x3, y3, z3);
  }
};

template< typename fixnum >
struct mnt4g2_pq_plus {
    __device__ void operator()(fixnum mod, fixnum x10, fixnum x11, fixnum y10, fixnum y11, fixnum z10, fixnum z11, fixnum x20, fixnum x21, fixnum y20, fixnum y21, fixnum z20, fixnum z21, fixnum &x30, fixnum &x31, fixnum &y30, fixnum &y31, fixnum &z30, fixnum &z31) {
        typedef mnt4_g2<fixnum> mnt4g2;
        mnt4g2::pq_plus(mod, x10, x11, y10, y11, z10, z11, x20, x21, y20, y21, z20, z21, x30, x31, y30, y31, z30, z31);
  }
};

template< typename fixnum >
struct mnt6g1_pq_plus {
    __device__ void operator()(fixnum mod, fixnum x1, fixnum y1, fixnum z1, fixnum x2, fixnum y2, fixnum z2, fixnum &x3, fixnum &y3, fixnum &z3) {
        typedef mnt6_g1<fixnum> mnt6g1;
        mnt6g1::pq_plus(mod, x1, y1, z1, x2, y2, z2, x3, y3, z3);
  }
};

template< typename fixnum >
struct mnt6g2_pq_plus {
    __device__ void operator()(fixnum mod, fixnum x1, fixnum y1, fixnum z1, fixnum x2, fixnum y2, fixnum z2, fixnum &x3, fixnum &y3, fixnum &z3) {
        typedef mnt6_g2<fixnum> mnt6g2;
        mnt6g2::pq_plus(mod, x1, y1, z1, x2, y2, z2, x3, y3, z3);
  }
};

template< typename fixnum >
struct mnt4g1_calc_np {
    __device__ void operator()(fixnum mod, fixnum w, fixnum x1, fixnum y1, fixnum z1, fixnum &x3, fixnum &y3, fixnum &z3) {
    typedef modnum_monty_cios<fixnum> modnum;
    typedef mnt4_g1<fixnum> mnt4g1;
    modnum m(mod);
    fixnum rx, ry, rz;
    fixnum tempw = w;
    int i = 24*32 - 1;
    bool found_one = false;
    int count = 0;
#if 0
    if (threadIdx.x > 23) {
        dump(w, 24);
        dump(x1, 24);
        dump(y1, 24);
        dump(z1, 24);
    }
#endif
    //while(fixnum::cmp(tempw, fixnum::zero()) && i >= 0) {
    while(i >= 0) {
        size_t value = fixnum::get(tempw, i/32);
#if 0
        if (threadIdx.x > 23) {
            printf("i %d value[%d] %x\n", i, i/32, value);
        }
#endif
        if (found_one) {
            mnt4g1::p_double(mod, mod, rx, ry, rz, rx, ry, rz);
#if 0
            if (threadIdx.x > 23) {
            printf("double result\n");
            dump(rx, 24);
            }
#endif
        }
        if ((value)&(1<<i%32)) {
            if (found_one == false) {
                rx = x1;
                ry = y1;
                rz = z1;
            } else {
                mnt4g1::pq_plus(mod, rx, ry, rz, x1, y1, z1, rx, ry, rz);
            }
#if 0
            if (threadIdx.x > 23) {
            printf("add result\n");
            dump(rx, 24);
            }
#endif
            found_one = true;
        }
        i --;
        count ++;
#if 0
        if (threadIdx.x > 23) {
        //if (count >20) break;
        }
#endif
    }
    x3 = rx;
    y3 = ry;
    z3 = rz;
#if 0
    printf("final result\n");
    dump(x3, 24);
    dump(y3, 24);
    dump(z3, 24);
#endif
  }
};

template< typename fixnum >
struct mnt4g2_calc_np {
    __device__ void operator()(fixnum mod, fixnum w, fixnum x10, fixnum x11, fixnum y10, fixnum y11, fixnum z10, fixnum z11, fixnum &x30, fixnum &x31, fixnum &y30, fixnum &y31, fixnum &z30, fixnum &z31) {
    typedef modnum_monty_cios<fixnum> modnum;
    typedef mnt4_g2<fixnum> mnt4g2;
    modnum m(mod);
    fixnum tempw = w;
    int i = 24*32 - 1;
    bool found_one = false;
    int count = 0;
    while(i >= 0) {
        size_t value = fixnum::get(tempw, i/32);
        //printf("value[%d] is %x\n", i, value);
        if (found_one) {
            mnt4g2::p_double(mod, x30, x31, y30, y31, z30, z31, x30, x31, y30, y31, z30, z31);
        }
        if ((value)&(1<<i%32)) {
            if (found_one == false) {
                x30 = x10;
                x31 = x11;
                y30 = y10;
                y31 = y11;
                z30 = z10;
                z31 = z11;
            } else {
                mnt4g2::pq_plus(mod, x30, x31, y30, y31, z30, z31, x10, x11, y10, y11, z10, z11, x30, x31, y30, y31, z30, z31);
            }
            found_one = true;
        }
        i --;
        count ++;
        //if (count >= 50) break;
    }
  }
};

template< typename fixnum >
struct mnt6g1_calc_np {
    __device__ void operator()(fixnum mod, fixnum w, fixnum x1, fixnum y1, fixnum z1, fixnum &x3, fixnum &y3, fixnum &z3) {
    typedef modnum_monty_cios<fixnum> modnum;
    typedef mnt6_g1<fixnum> mnt6g1;
    modnum m(mod);
    fixnum rx, ry, rz;
    fixnum tempw = w;
    int i = 24*32 - 1;
    bool found_one = false;
    int count = 0;
    while(i >= 0) {
        size_t value = fixnum::get(tempw, i/32);
        if (found_one) {
            mnt6g1::p_double(mod, mod, rx, ry, rz, rx, ry, rz);
        }
        if ((value)&(1<<i%32)) {
            if (found_one == false) {
                rx = x1;
                ry = y1;
                rz = z1;
            } else {
                mnt6g1::pq_plus(mod, rx, ry, rz, x1, y1, z1, rx, ry, rz);
            }
            found_one = true;
        }
        i --;
        count ++;
    }
    x3 = rx;
    y3 = ry;
    z3 = rz;
  }
};

template< typename fixnum >
struct mnt6g2_calc_np {
    __device__ void operator()(fixnum mod, fixnum w, fixnum x1, fixnum y1, fixnum z1, fixnum &x3, fixnum &y3, fixnum &z3) {
    typedef modnum_monty_cios<fixnum> modnum;
    typedef mnt6_g2<fixnum> mnt6g2;
    modnum m(mod);
    fixnum rx, ry, rz;
    fixnum tempw = w;
    int i = 24*32 - 1;
    bool found_one = false;
    int count = 0;
    while(i >= 0) {
        size_t value = fixnum::get(tempw, i/32);
        if (found_one) {
            mnt6g2::p_double(mod, mod, rx, ry, rz, rx, ry, rz);
        }
        if ((value)&(1<<i%32)) {
            if (found_one == false) {
                rx = x1;
                ry = y1;
                rz = z1;
            } else {
                mnt6g2::pq_plus(mod, rx, ry, rz, x1, y1, z1, rx, ry, rz);
            }
            found_one = true;
        }
        i --;
        count ++;
    }
    x3 = rx;
    y3 = ry;
    z3 = rz;
  }
};


int mnt4_g1_pq_plus(int n, uint8_t* x1, uint8_t* y1, uint8_t* z1, uint8_t* x2, uint8_t* y2, uint8_t* z2, uint8_t *x3, uint8_t *y3, uint8_t *z3) {
    typedef warp_fixnum<96, u32_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;
    fixnum_array *x1in, *y1in, *z1in, *x2in, *y2in, *z2in;
    fixnum_array *rx3, *ry3, *rz3;
    int fn_bytes = 96;
    int step_bytes = n * fn_bytes;
    uint8_t *modulus_bytes = new uint8_t[step_bytes];
    // mnt4 q
    for(int i = 0; i < n; i++) {
        memcpy(modulus_bytes + i*MNT_SIZE, mnt4_modulus, MNT_SIZE);
    }
    auto modulus4 = fixnum_array::create(modulus_bytes, step_bytes, MNT_SIZE);
    
    x1in = fixnum_array::create(x1, step_bytes, fn_bytes);
    y1in = fixnum_array::create(y1, step_bytes, fn_bytes);
    z1in = fixnum_array::create(z1, step_bytes, fn_bytes);
    x2in = fixnum_array::create(x2, step_bytes, fn_bytes);
    y2in = fixnum_array::create(y2, step_bytes, fn_bytes);
    z2in = fixnum_array::create(z2, step_bytes, fn_bytes);

    rx3 = fixnum_array::create(n);
    ry3 = fixnum_array::create(n);
    rz3 = fixnum_array::create(n);
    fixnum_array::template map<mnt4g1_pq_plus>(modulus4, x1in, y1in, z1in, x2in, y2in, z2in, rx3, ry3, rz3);

    int size = n; 
    rx3->retrieve_all(x3, step_bytes, &size);
    ry3->retrieve_all(y3, step_bytes, &size);
    rz3->retrieve_all(z3, step_bytes, &size);
   
    delete x1in; 
    delete y1in; 
    delete z1in; 
    delete x2in; 
    delete y2in; 
    delete z2in; 
    delete rx3;
    delete ry3;
    delete rz3;
    delete modulus4;
    delete modulus_bytes;
    return 0;
}

inline void do_sigma(int nelts, int type, uint8_t *x, uint8_t *y, uint8_t *z, uint8_t *rx, uint8_t *ry, uint8_t *rz) {
    typedef warp_fixnum<96, u32_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;
    fixnum_array *x1in, *y1in, *z1in, *x2in, *y2in, *z2in;
    fixnum_array *rx3, *ry3, *rz3;

    int half_bytes = MNT_SIZE*nelts/2;
    uint8_t *modulus_bytes = new uint8_t[half_bytes];
    // mnt4 q
    for(int i = 0; i < nelts/2; i++) {
        memcpy(modulus_bytes + i*MNT_SIZE, mnt4_modulus, MNT_SIZE);
    }
    auto modulus4 = fixnum_array::create(modulus_bytes, half_bytes, MNT_SIZE);
    x1in = fixnum_array::create(x, half_bytes, MNT_SIZE);
    y1in = fixnum_array::create(y, half_bytes, MNT_SIZE);
    z1in = fixnum_array::create(z, half_bytes, MNT_SIZE);
    x2in = fixnum_array::create(x + half_bytes, half_bytes, MNT_SIZE);
    y2in = fixnum_array::create(y + half_bytes, half_bytes, MNT_SIZE);
    z2in = fixnum_array::create(z + half_bytes, half_bytes, MNT_SIZE);

    rx3 = fixnum_array::create(nelts/2);
    ry3 = fixnum_array::create(nelts/2);
    rz3 = fixnum_array::create(nelts/2);
    fixnum_array::template map<mnt4g1_pq_plus>(modulus4, x1in, y1in, z1in, x2in, y2in, z2in, rx3, ry3, rz3);
    
    int size = nelts/2;
    rx3->retrieve_all(rx, half_bytes, &size);
    ry3->retrieve_all(ry, half_bytes, &size);
    rz3->retrieve_all(rz, half_bytes, &size);
    delete x1in;
    delete y1in;
    delete z1in;
    delete x2in;
    delete y2in;
    delete z2in;
    delete rx3;
    delete ry3;
    delete rz3;
    delete modulus4;
    delete modulus_bytes;
}

int do_calc_np_sigma(int nelts, uint8_t* scalar, uint8_t* x1, uint8_t* y1, uint8_t* z1, uint8_t *x3, uint8_t *y3, uint8_t *z3) {
    clock_t start = clock();
    typedef warp_fixnum<96, u32_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;
    printf("calc do_calc_np_sigma\n");
    printf("nelts %d\n", nelts);
    int step = nelts;
    int size = nelts;
    int DATA_SIZE = 96;
    int fn_bytes = DATA_SIZE;
    int step_bytes = fn_bytes * step;
    uint8_t *x1bytes = x1;
    uint8_t *y1bytes = y1;
    uint8_t *z1bytes = z1;
    uint8_t *x3bytes = new uint8_t[step_bytes];
    uint8_t *y3bytes = new uint8_t[step_bytes];
    uint8_t *z3bytes = new uint8_t[step_bytes];
    fixnum_array *dx3, *dy3, *dz3, *x1in, *y1in, *z1in;
    fixnum_array *x2in, *y2in, *z2in;
    uint8_t *modulus_bytes = new uint8_t[step_bytes];
    // mnt4 q
    memset(modulus_bytes, 0x0, step_bytes);
    for(int i = 0; i < step; i++) {
        memcpy(modulus_bytes + i*fn_bytes, mnt4_modulus, fn_bytes);
    }
    auto modulus4 = fixnum_array::create(modulus_bytes, step_bytes, fn_bytes);

    // mnt6 q
    memset(modulus_bytes, 0x0, step_bytes);
    for(int i = 0; i < step; i++) {
        memcpy(modulus_bytes + i*fn_bytes, mnt6_modulus, fn_bytes);
    }
    auto modulus6 = fixnum_array::create(modulus_bytes, step_bytes, fn_bytes);

    // mnt4 a
    memset(modulus_bytes, 0x0, step_bytes);
    for(int i = 0; i < step; i++) {
        memcpy(modulus_bytes + i*fn_bytes, mnt4_a, fn_bytes);
    }
    auto mnt4a = fixnum_array::create(modulus_bytes, step_bytes, fn_bytes);

    // scaler
    auto modulusw = fixnum_array::create(scalar, step_bytes, fn_bytes);
    
    // sigma result
    fixnum_array *rx3, *ry3, *rz3;
    int got_result = false;

    for (int i = 0; i < nelts; i+=step) {
        dx3 = fixnum_array::create(step);
        dy3 = fixnum_array::create(step);
        dz3 = fixnum_array::create(step);
        x1in = fixnum_array::create(x1bytes, step_bytes, fn_bytes);
        y1in = fixnum_array::create(y1bytes, step_bytes, fn_bytes);
        z1in = fixnum_array::create(z1bytes, step_bytes, fn_bytes);
        fixnum_array::template map<mnt4g1_calc_np>(modulus4, modulusw, x1in, y1in, z1in, dx3, dy3, dz3);

        dx3->retrieve_all(x3bytes, step_bytes, &size);
        dy3->retrieve_all(y3bytes, step_bytes, &size);
        dz3->retrieve_all(z3bytes, step_bytes, &size);
#ifdef PARALLEL_SIGMA
        int start = nelts%2;
        int rnelts = nelts - start;
        uint8_t *rx, *ry, *rz;
        rx = new uint8_t[MNT_SIZE*rnelts/2];
        ry = new uint8_t[MNT_SIZE*rnelts/2];
        rz = new uint8_t[MNT_SIZE*rnelts/2];
        while(rnelts > 1) {
            do_sigma(rnelts, 1, x3bytes + start*MNT_SIZE, y3bytes + start*MNT_SIZE, z3bytes + start*MNT_SIZE, rx, ry, rz);
            rnelts = rnelts >> 1;
            memcpy(x3bytes + start*MNT_SIZE, rx, rnelts*MNT_SIZE);
            memcpy(y3bytes + start*MNT_SIZE, ry, rnelts*MNT_SIZE);
            memcpy(z3bytes + start*MNT_SIZE, rz, rnelts*MNT_SIZE);
            if (rnelts > 1 && rnelts%2) {
                if (start == 0) {
                    start = 1;
                    rnelts -= 1;
                } else {
                    start = 0;
                    rnelts += 1;
                }
            }
        }
        delete rx;
        delete ry;
        delete rz;
        if (start == 1) {
            // add the first element
            x2in = fixnum_array::create(x3bytes, fn_bytes, fn_bytes);
            y2in = fixnum_array::create(y3bytes, fn_bytes, fn_bytes);
            z2in = fixnum_array::create(z3bytes, fn_bytes, fn_bytes);
            rx3 = fixnum_array::create(x3bytes + fn_bytes, fn_bytes, fn_bytes);
            ry3 = fixnum_array::create(y3bytes + fn_bytes, fn_bytes, fn_bytes);
            rz3 = fixnum_array::create(z3bytes + fn_bytes, fn_bytes, fn_bytes);
            fixnum_array::template map<mnt4g1_pq_plus>(modulus4, x2in, y2in, z2in, rx3, ry3, rz3, rx3, ry3, rz3);
            delete x2in;
            delete y2in;
            delete z2in;
        } else {
            memcpy(x3, x3bytes, fn_bytes);
            memcpy(y3, y3bytes, fn_bytes);
            memcpy(z3, z3bytes, fn_bytes);
            got_result = true;
        }
#else
        bool result_set = false;
        // start add from second element
        int start = 1;
        if (i == 0) {
            rx3 = fixnum_array::create(x3bytes + start * fn_bytes, fn_bytes, fn_bytes);
            ry3 = fixnum_array::create(y3bytes + start * fn_bytes, fn_bytes, fn_bytes);
            rz3 = fixnum_array::create(z3bytes + start * fn_bytes, fn_bytes, fn_bytes);
            result_set = true;
        }
        int k = 0;
        if (result_set && i == 0) {
            k = start + 1;
        }
        for (; k < step; k ++)
        {
            x2in = fixnum_array::create(x3bytes + k * fn_bytes, fn_bytes, fn_bytes);
            y2in = fixnum_array::create(y3bytes + k * fn_bytes, fn_bytes, fn_bytes);
            z2in = fixnum_array::create(z3bytes + k * fn_bytes, fn_bytes, fn_bytes);
            fixnum_array::template map<mnt4g1_pq_plus>(modulus4, rx3, ry3, rz3, x2in, y2in, z2in, rx3, ry3, rz3);
            delete x2in;
            delete y2in;
            delete z2in;
        }
        // add the first element
        x2in = fixnum_array::create(x3bytes, fn_bytes, fn_bytes);
        y2in = fixnum_array::create(y3bytes, fn_bytes, fn_bytes);
        z2in = fixnum_array::create(z3bytes, fn_bytes, fn_bytes);
        fixnum_array::template map<mnt4g1_pq_plus>(modulus4, x2in, y2in, z2in, rx3, ry3, rz3, rx3, ry3, rz3);
        delete x2in;
        delete y2in;
        delete z2in;
#endif
        delete x1in;
        delete y1in;
        delete z1in;
        delete dx3;
        delete dy3;
        delete dz3;
    }
    if (!got_result) {
        size = 1;
        rx3->retrieve_all(x3, fn_bytes, &size);
        ry3->retrieve_all(y3, fn_bytes, &size);
        rz3->retrieve_all(z3, fn_bytes, &size);
        delete rx3;
        delete ry3;
        delete rz3;
    }
    delete x3bytes;
    delete y3bytes;
    delete z3bytes;
    delete modulus_bytes;

    printf("final result");
    printf("\nx3:");
    for (int k = fn_bytes-1; k >= 0; k--) {
        printf("%02x", x3[k]);
    }
    printf("\ny3:");
    for (int k = fn_bytes-1; k >= 0; k--) {
        printf("%02x", y3[k]);
    }
    printf("\nz3:");
    for (int k = fn_bytes-1; k >= 0; k--) {
       printf("%02x", z3[k]);
    }
    printf("\n");
    clock_t diff = clock() - start;
    printf("cost time %ld\n", diff);
    return 0;
}

inline void do_sigma(int nelts, int type, uint8_t *x0, uint8_t *x1, uint8_t *y0, uint8_t *y1, uint8_t *z0, uint8_t *z1, uint8_t *rx0, uint8_t *rx1, uint8_t *ry0, uint8_t *ry1, uint8_t *rz0, uint8_t *rz1) {
    typedef warp_fixnum<96, u32_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;
    fixnum_array *x10in, *x11in, *y10in, *y11in, *z10in, *z11in, *x20in, *x21in, *y20in, *y21in, *z20in, *z21in;
    fixnum_array *rx30, *rx31, *ry30, *ry31, *rz30, *rz31;

    int half_bytes = MNT_SIZE*nelts/2;
    uint8_t *modulus_bytes = new uint8_t[half_bytes];
    // mnt4 q
    for(int i = 0; i < nelts/2; i++) {
        memcpy(modulus_bytes + i*MNT_SIZE, mnt4_modulus, MNT_SIZE);
    }
    auto modulus4 = fixnum_array::create(modulus_bytes, half_bytes, MNT_SIZE);
    x10in = fixnum_array::create(x0, half_bytes, MNT_SIZE);
    x11in = fixnum_array::create(x1, half_bytes, MNT_SIZE);
    y10in = fixnum_array::create(y0, half_bytes, MNT_SIZE);
    y11in = fixnum_array::create(y1, half_bytes, MNT_SIZE);
    z10in = fixnum_array::create(z0, half_bytes, MNT_SIZE);
    z11in = fixnum_array::create(z1, half_bytes, MNT_SIZE);
    x20in = fixnum_array::create(x0 + half_bytes, half_bytes, MNT_SIZE);
    x21in = fixnum_array::create(x1 + half_bytes, half_bytes, MNT_SIZE);
    y20in = fixnum_array::create(y0 + half_bytes, half_bytes, MNT_SIZE);
    y21in = fixnum_array::create(y1 + half_bytes, half_bytes, MNT_SIZE);
    z20in = fixnum_array::create(z0 + half_bytes, half_bytes, MNT_SIZE);
    z21in = fixnum_array::create(z1 + half_bytes, half_bytes, MNT_SIZE);

    rx30 = fixnum_array::create(nelts/2);
    rx31 = fixnum_array::create(nelts/2);
    ry30 = fixnum_array::create(nelts/2);
    ry31 = fixnum_array::create(nelts/2);
    rz30 = fixnum_array::create(nelts/2);
    rz31 = fixnum_array::create(nelts/2);
    fixnum_array::template map<mnt4g2_pq_plus>(modulus4, x10in, x11in, y10in, y11in, z10in, z11in, x20in, x21in, y20in, y21in, z20in, z21in, rx30, rx31, ry30, ry31, rz30, rz31);
    
    int size = nelts/2;
    rx30->retrieve_all(rx0, half_bytes, &size);
    rx31->retrieve_all(rx1, half_bytes, &size);
    ry30->retrieve_all(ry0, half_bytes, &size);
    ry31->retrieve_all(ry1, half_bytes, &size);
    rz30->retrieve_all(rz0, half_bytes, &size);
    rz31->retrieve_all(rz1, half_bytes, &size);
    delete x10in;
    delete x11in;
    delete y10in;
    delete y11in;
    delete z10in;
    delete z11in;
    delete x20in;
    delete x21in;
    delete y20in;
    delete y21in;
    delete z20in;
    delete z21in;
    delete rx30;
    delete rx31;
    delete ry30;
    delete ry31;
    delete rz30;
    delete rz31;
    delete modulus4;
    delete modulus_bytes;
}

void print_g2(uint8_t *x30, uint8_t *x31, uint8_t *y30, uint8_t *y31, uint8_t *z30, uint8_t *z31) {
    int fn_bytes = 96;
    printf("\nx3:");
    for (int k = fn_bytes-1; k >= 0; k--) {
        printf("%02x", x30[k]);
    }
    printf(",");
    for (int k = fn_bytes-1; k >= 0; k--) {
        printf("%02x", x31[k]);
    }
    printf("\ny3:");
    for (int k = fn_bytes-1; k >= 0; k--) {
        printf("%02x", y30[k]);
    }
    printf(",");
    for (int k = fn_bytes-1; k >= 0; k--) {
        printf("%02x", y31[k]);
    }
    printf("\nz3:");
    for (int k = fn_bytes-1; k >= 0; k--) {
       printf("%02x", z30[k]);
    }
    printf(",");
    for (int k = fn_bytes-1; k >= 0; k--) {
       printf("%02x", z31[k]);
    }
    printf("\n");

}

int do_calc_np_sigma_mnt4_g2(int nelts, uint8_t * scalar, uint8_t* x10, uint8_t* x11, uint8_t* y10, uint8_t* y11, uint8_t* z10, uint8_t* z11, uint8_t *x30, uint8_t *x31, uint8_t *y30, uint8_t *y31, uint8_t *z30, uint8_t *z31) {
    clock_t start = clock();
    typedef warp_fixnum<96, u32_fixnum> fixnum;
    typedef fixnum_array<fixnum> fixnum_array;
    printf("calc do_calc_np_sigma_mnt4_g2\n");
    printf("nelts %d\n", nelts);
    // test
#if 0
    nelts = 1;
    int sn = 96;
    x10 += sn;
    x11 += sn;
    y10 += sn;
    y11 += sn;
    z10 += sn;
    z11 += sn;
    scalar += sn;
#endif
    int step = nelts;
    int size = nelts;
    int DATA_SIZE = 96;
    int fn_bytes = DATA_SIZE;
    int step_bytes = fn_bytes * step;
    uint8_t *x10bytes = x10;
    uint8_t *x11bytes = x11;
    uint8_t *y10bytes = y10;
    uint8_t *y11bytes = y11;
    uint8_t *z10bytes = z10;
    uint8_t *z11bytes = z11;
    uint8_t *x30bytes = new uint8_t[step_bytes];
    uint8_t *x31bytes = new uint8_t[step_bytes];
    uint8_t *y30bytes = new uint8_t[step_bytes];
    uint8_t *y31bytes = new uint8_t[step_bytes];
    uint8_t *z30bytes = new uint8_t[step_bytes];
    uint8_t *z31bytes = new uint8_t[step_bytes];
    fixnum_array *dx30, *dx31, *dy30, *dy31, *dz30, *dz31, *x10in, *x11in, *y10in, *y11in, *z10in, *z11in;
    fixnum_array *x20in, *x21in, *y20in, *y21in, *z20in, *z21in;
    uint8_t *modulus_bytes = new uint8_t[step_bytes];
    // mnt4 q
    memset(modulus_bytes, 0x0, step_bytes);
    for(int i = 0; i < step; i++) {
        memcpy(modulus_bytes + i*fn_bytes, mnt4_modulus, fn_bytes);
    }
    auto modulus4 = fixnum_array::create(modulus_bytes, step_bytes, fn_bytes);

    // mnt6 q
    memset(modulus_bytes, 0x0, step_bytes);
    for(int i = 0; i < step; i++) {
        memcpy(modulus_bytes + i*fn_bytes, mnt6_modulus, fn_bytes);
    }
    auto modulus6 = fixnum_array::create(modulus_bytes, step_bytes, fn_bytes);

    // mnt4 a
    memset(modulus_bytes, 0x0, step_bytes);
    for(int i = 0; i < step; i++) {
        memcpy(modulus_bytes + i*fn_bytes, mnt4_a, fn_bytes);
    }
    auto mnt4a = fixnum_array::create(modulus_bytes, step_bytes, fn_bytes);

    // scalar
    auto modulusw = fixnum_array::create(scalar, step_bytes, fn_bytes);
    
    // sigma result
    fixnum_array *rx30, *rx31, *ry30, *ry31, *rz30, *rz31;
    int got_result = false;

#if 0
    printf("x10:\n");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", x10bytes[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\nx11:");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", x11bytes[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\ny10:");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", y10bytes[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\ny11:");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", y11bytes[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\nz10:");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", z10bytes[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\nz11:");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", z11bytes[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\nscalar:");
    for (int i = 0; i < step_bytes; i ++) {
        printf("%02x", scalar[i]); 
        if ((i+1) % 96 == 0) {
            printf("\t");
        }
    }
    printf("\n");
#endif
    for (int i = 0; i < nelts; i+=step) {
        dx30 = fixnum_array::create(step);
        dx31 = fixnum_array::create(step);
        dy30 = fixnum_array::create(step);
        dy31 = fixnum_array::create(step);
        dz30 = fixnum_array::create(step);
        dz31 = fixnum_array::create(step);
        x10in = fixnum_array::create(x10bytes, step_bytes, fn_bytes);
        x11in = fixnum_array::create(x11bytes, step_bytes, fn_bytes);
        y10in = fixnum_array::create(y10bytes, step_bytes, fn_bytes);
        y11in = fixnum_array::create(y11bytes, step_bytes, fn_bytes);
        z10in = fixnum_array::create(z10bytes, step_bytes, fn_bytes);
        z11in = fixnum_array::create(z11bytes, step_bytes, fn_bytes);
        fixnum_array::template map<mnt4g2_calc_np>(modulus4, modulusw, x10in, x11in, y10in, y11in, z10in, z11in, dx30, dx31, dy30, dy31, dz30, dz31);

        dx30->retrieve_all(x30bytes, step_bytes, &size);
        dx31->retrieve_all(x31bytes, step_bytes, &size);
        dy30->retrieve_all(y30bytes, step_bytes, &size);
        dy31->retrieve_all(y31bytes, step_bytes, &size);
        dz30->retrieve_all(z30bytes, step_bytes, &size);
        dz31->retrieve_all(z31bytes, step_bytes, &size);
        delete x10in;
        delete x11in;
        delete y10in;
        delete y11in;
        delete z10in;
        delete z11in;
        delete dx30;
        delete dx31;
        delete dy30;
        delete dy31;
        delete dz30;
        delete dz31;
#ifdef PARALLEL_SIGMA
        int start = nelts%2;
        int rnelts = nelts - start;
        uint8_t *rx0, *rx1, *ry0, *ry1, *rz0, *rz1;
        rx0 = new uint8_t[MNT_SIZE*rnelts/2];
        rx1 = new uint8_t[MNT_SIZE*rnelts/2];
        ry0 = new uint8_t[MNT_SIZE*rnelts/2];
        ry1 = new uint8_t[MNT_SIZE*rnelts/2];
        rz0 = new uint8_t[MNT_SIZE*rnelts/2];
        rz1 = new uint8_t[MNT_SIZE*rnelts/2];
        while(rnelts > 1) {
            do_sigma(rnelts, 1, x30bytes + start*MNT_SIZE, x31bytes + start*MNT_SIZE, y30bytes + start*MNT_SIZE, y31bytes + start*MNT_SIZE, z30bytes + start*MNT_SIZE, z31bytes + start*MNT_SIZE, rx0, rx1, ry0, ry1, rz0, rz1);
            rnelts = rnelts >> 1;
            memcpy(x30bytes + start*MNT_SIZE, rx0, rnelts*MNT_SIZE);
            memcpy(x31bytes + start*MNT_SIZE, rx1, rnelts*MNT_SIZE);
            memcpy(y30bytes + start*MNT_SIZE, ry0, rnelts*MNT_SIZE);
            memcpy(y31bytes + start*MNT_SIZE, ry1, rnelts*MNT_SIZE);
            memcpy(z30bytes + start*MNT_SIZE, rz0, rnelts*MNT_SIZE);
            memcpy(z31bytes + start*MNT_SIZE, rz1, rnelts*MNT_SIZE);
            if (rnelts > 1 && rnelts%2) {
                if (start == 0) {
                    start = 1;
                    rnelts -= 1;
                } else {
                    start = 0;
                    rnelts += 1;
                }
            }
        }
        delete rx0;
        delete rx1;
        delete ry0;
        delete ry1;
        delete rz0;
        delete rz1;
        if (start == 1) {
            // add the first element
            x20in = fixnum_array::create(x30bytes, fn_bytes, fn_bytes);
            x21in = fixnum_array::create(x31bytes, fn_bytes, fn_bytes);
            y20in = fixnum_array::create(y30bytes, fn_bytes, fn_bytes);
            y21in = fixnum_array::create(y31bytes, fn_bytes, fn_bytes);
            z20in = fixnum_array::create(z30bytes, fn_bytes, fn_bytes);
            z21in = fixnum_array::create(z31bytes, fn_bytes, fn_bytes);
            rx30 = fixnum_array::create(x30bytes + fn_bytes, fn_bytes, fn_bytes);
            rx31 = fixnum_array::create(x31bytes + fn_bytes, fn_bytes, fn_bytes);
            ry30 = fixnum_array::create(y30bytes + fn_bytes, fn_bytes, fn_bytes);
            ry31 = fixnum_array::create(y31bytes + fn_bytes, fn_bytes, fn_bytes);
            rz30 = fixnum_array::create(z30bytes + fn_bytes, fn_bytes, fn_bytes);
            rz31 = fixnum_array::create(z31bytes + fn_bytes, fn_bytes, fn_bytes);
            fixnum_array::template map<mnt4g2_pq_plus>(modulus4, rx30, rx31, ry30, ry31, rz30, rz31, x20in, x21in, y20in, y21in, z20in, z21in, rx30, rx31, ry30, ry31, rz30, rz31);
            delete x20in;
            delete x21in;
            delete y20in;
            delete y21in;
            delete z20in;
            delete z21in;
        } else {
            memcpy(x30, x30bytes, fn_bytes);
            memcpy(x31, x31bytes, fn_bytes);
            memcpy(y30, y30bytes, fn_bytes);
            memcpy(y31, y31bytes, fn_bytes);
            memcpy(z30, z30bytes, fn_bytes);
            memcpy(z31, z31bytes, fn_bytes);
            got_result = true;
        }
#else
        bool result_set = false;
        // start add from second element
        int start = 1;
        if (i == 0) {
            //print_g2(x30bytes + start * fn_bytes, x31bytes + start * fn_bytes, y30bytes + start * fn_bytes, y31bytes + start * fn_bytes, z30bytes + start * fn_bytes, z31bytes + start * fn_bytes);
            rx30 = fixnum_array::create(x30bytes + start * fn_bytes, fn_bytes, fn_bytes);
            rx31 = fixnum_array::create(x31bytes + start * fn_bytes, fn_bytes, fn_bytes);
            ry30 = fixnum_array::create(y30bytes + start * fn_bytes, fn_bytes, fn_bytes);
            ry31 = fixnum_array::create(y31bytes + start * fn_bytes, fn_bytes, fn_bytes);
            rz30 = fixnum_array::create(z30bytes + start * fn_bytes, fn_bytes, fn_bytes);
            rz31 = fixnum_array::create(z31bytes + start * fn_bytes, fn_bytes, fn_bytes);
            result_set = true;
        }
        int k = 0;
        if (result_set && i == 0) {
            k = start + 1;
        }
        for (; k < step; k ++)
        {
            //print_g2(x30bytes + k * fn_bytes, x31bytes + k * fn_bytes, y30bytes + k * fn_bytes, y31bytes + k * fn_bytes, z30bytes + k * fn_bytes, z31bytes + k * fn_bytes);
            x20in = fixnum_array::create(x30bytes + k * fn_bytes, fn_bytes, fn_bytes);
            x21in = fixnum_array::create(x31bytes + k * fn_bytes, fn_bytes, fn_bytes);
            y20in = fixnum_array::create(y30bytes + k * fn_bytes, fn_bytes, fn_bytes);
            y21in = fixnum_array::create(y31bytes + k * fn_bytes, fn_bytes, fn_bytes);
            z20in = fixnum_array::create(z30bytes + k * fn_bytes, fn_bytes, fn_bytes);
            z21in = fixnum_array::create(z31bytes + k * fn_bytes, fn_bytes, fn_bytes);
            fixnum_array::template map<mnt4g2_pq_plus>(modulus4, rx30, rx31, ry30, ry31, rz30, rz31, x20in, x21in, y20in, y21in, z20in, z21in, rx30, rx31, ry30, ry31, rz30, rz31);
            delete x20in;
            delete x21in;
            delete y20in;
            delete y21in;
            delete z20in;
            delete z21in;
#if 0
        rx30->retrieve_all(x30, fn_bytes, &size);
        rx31->retrieve_all(x31, fn_bytes, &size);
        ry30->retrieve_all(y30, fn_bytes, &size);
        ry31->retrieve_all(y31, fn_bytes, &size);
        rz30->retrieve_all(z30, fn_bytes, &size);
        rz31->retrieve_all(z31, fn_bytes, &size);
            print_g2(x30, x31, y30, y31, z30, z31);
#endif
        }
        // add the first element
        x20in = fixnum_array::create(x30bytes, fn_bytes, fn_bytes);
        x21in = fixnum_array::create(x31bytes, fn_bytes, fn_bytes);
        y20in = fixnum_array::create(y30bytes, fn_bytes, fn_bytes);
        y21in = fixnum_array::create(y31bytes, fn_bytes, fn_bytes);
        z20in = fixnum_array::create(z30bytes, fn_bytes, fn_bytes);
        z21in = fixnum_array::create(z31bytes, fn_bytes, fn_bytes);
        fixnum_array::template map<mnt4g2_pq_plus>(modulus4, rx30, rx31, ry30, ry31, rz30, rz31, x20in, x21in, y20in, y21in, z20in, z21in, rx30, rx31, ry30, ry31, rz30, rz31);
        delete x20in;
        delete x21in;
        delete y20in;
        delete y21in;
        delete z20in;
        delete z21in;
#endif
    }
    if (!got_result) {
        size = 1;
        rx30->retrieve_all(x30, fn_bytes, &size);
        rx31->retrieve_all(x31, fn_bytes, &size);
        ry30->retrieve_all(y30, fn_bytes, &size);
        ry31->retrieve_all(y31, fn_bytes, &size);
        rz30->retrieve_all(z30, fn_bytes, &size);
        rz31->retrieve_all(z31, fn_bytes, &size);
        delete rx30;
        delete rx31;
        delete ry30;
        delete ry31;
        delete rz30;
        delete rz31;
    }
    delete x30bytes;
    delete x31bytes;
    delete y30bytes;
    delete y31bytes;
    delete z30bytes;
    delete z31bytes;
    delete modulus_bytes;

    printf("mnt4_g2 final result");
    print_g2(x30, x31, y30, y31, z30, z31);
    clock_t diff = clock() - start;
    printf("cost time %ld\n", diff);
    return 0;
}
